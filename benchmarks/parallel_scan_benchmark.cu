#include <iostream>
#include <vector>
#include <mg_gpgpu_core/parallel_scan.h>

template <unsigned int ITERATIONS>
void bench_scan_algs()
{

    std::vector<uint32_t > data;
    std::vector<uint32_t > original;
    uint32_t size = rand() %(100000) ;

    data.resize(size);
    original.resize(size);
    for (int i =0 ; i <size; ++i)
    {
        data[i] = rand() % 2 + 1;
        original[i] = data[i];
    }
    auto ptr = data.data();
    uint32_t* in;
    uint32_t* out;

    hipMalloc( (void**)&in,  data.size()*sizeof(uint32_t));
    hipMalloc( (void**)&out,  data.size()*sizeof(uint32_t));
    hipMemcpy( in, data.data(),data.size()*sizeof(uint32_t), hipMemcpyHostToDevice );

    //creating timing stuff
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    
    float milliseconds = 0;

    hipEventRecord(start);
    for (int i =0; i< ITERATIONS; ++i)
    {
        auto cudares = mg_gpgpu::parallel_scan_hillis_steel<uint32_t>(in,out, size);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout<<"parallel_scan hilliss_steel took: "<<
                (milliseconds/ (static_cast<float>(ITERATIONS)))<< " ms"<<std::endl;

    hipFree(in);
    hipFree(out);
}
