#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <mg_gpgpu_core/parallel_scan.h>

template <unsigned int ITERATIONS>
void bench_scan_algs()
{

    std::cout<<"================================================"<<std::endl;
    std::cout<<"                  SCAN                          "<<std::endl;
    std::cout<<"================================================"<<std::endl;
    std::vector<uint32_t > data;
    std::vector<uint32_t > original;
    uint32_t size = rand() %(1000000) ;

    data.resize(size);
    original.resize(size);
    for (int i =0 ; i <size; ++i)
    {
        data[i] = rand() % 2 + 1;
        original[i] = data[i];
    }
    auto ptr = data.data();
    uint32_t* in;
    uint32_t* out;

    hipMalloc( (void**)&in,  data.size()*sizeof(uint32_t));
    hipMalloc( (void**)&out,  data.size()*sizeof(uint32_t));
    hipMemcpy( in, data.data(),data.size()*sizeof(uint32_t), hipMemcpyHostToDevice );

    //creating timing stuff
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    
    float milliseconds = 0;

    hipEventRecord(start);
    for (int i =0; i< ITERATIONS; ++i)
    {
        auto cudares = mg_gpgpu::parallel_scan_hillis_steel<uint32_t>(in,out, size);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout<<"parallel_scan hilliss_steel took: "<<
                (milliseconds/ (static_cast<float>(ITERATIONS)))<< " ms"<<std::endl;


    milliseconds = 0;
    ////computing the wanted blocks
    uint32_t* d_intermediate;
    uint32_t threads = 512;
    uint32_t blocks = ((size%threads) != 0)?(size/threads) +1 : (size/threads);
    //here we have an extra one which will be our atomic value for blocks
    if (blocks == 0)
    {blocks =1;}

    //compute_blocks(threads, blocks,count);
    gpuErrchkDebug(hipMalloc( (void**)&d_intermediate,  (blocks + 1)*sizeof(uint32_t)));

    constexpr uint32_t SENTINEL = std::numeric_limits<uint32_t>::max();
    hipEventRecord(start);
    for (int i =0; i< ITERATIONS; ++i)
    {
        //mg_gpgpu::parallel_stream_scan<uint32_t,SENTINEL>(in,d_intermediate, size);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout<<"parallel stream scan took: "<<
                (milliseconds/ (static_cast<float>(ITERATIONS)))<< " ms"<<std::endl;



    //hipFree(in);
    //hipFree(out);
}
